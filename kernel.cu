#include "hip/hip_runtime.h"
﻿
#include "kernel.h"

#include <>
#include <hip/hip_runtime.h>

/*
Функція ядра заповнює нову матрицю у відповідності до блоків-сегментів на 
які поділена матриця вхідних даних.
*/
__global__ 
void MatrixBlockMask(int* c, const int* a, const int* b, int w, int h)
{


	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;

	c[row * w + col] = blockIdx.y + blockIdx.x + (blockIdx.y * (blockDim.x - 1));
}

/*
Задано розмір блока-сегмента у 4 колонки та 8 рядків. За таких розмірів сегментів,
матриця вхідних даних, що має розміри 16х16, буде сегментована у 8 блоків-сегметів.
*/
void matrix_block_mask(int* c, const int* a, const int *b, int w, int h)
{
	int N = w * h * sizeof(int);
	int *d_a, *d_b, *d_c;

	hipMalloc(&d_a, N);
	hipMalloc(&d_b, N);
	hipMalloc(&d_c, N);

	hipError_t error;

	hipMemcpy(d_a, a, N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N, hipMemcpyHostToDevice);

	dim3 dimBlock(4, 8);
	dim3 dimGrid(w / dimBlock.x, h / dimBlock.y);

	MatrixBlockMask<<<dimGrid, dimBlock>>> (d_c, d_a, d_b, w , h);
	
	error = hipDeviceSynchronize();

	hipMemcpy(c, d_c, N, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
