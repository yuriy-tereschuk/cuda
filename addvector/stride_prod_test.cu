#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "tests.h"
#include "tools.h"

#define THREADS 512

using namespace std;

__global__
void stride_prod(const int* a, const int* b, int* prod, int elements)
{
  __shared__ int temp[THREADS];

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  temp[threadIdx.x] = a[idx] + b[idx];

  __syncthreads();

  int sum = 0;
  for (int i = 0; i < elements; i += gridDim.x * blockDim.x)
  {
    sum += temp[i];
  }
  atomicAdd(prod, sum);
}

void stride_prod_test()
{

  hipError_t err = hipSuccess;

  int *h_a, *h_b, *h_prod;
  int *d_a, *d_b, *d_prod;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int elements = 5 << 10;

  h_a = new int[elements];
  h_b = new int[elements];
  h_prod = new int;

  init(h_a, elements);
  init(h_b, elements);
  *h_prod = 0;

  std::cout << "Host prod: " << host_prod(h_a, h_b, elements) << std::endl;

  hipMalloc((void**) &d_a, sizeof(int) * elements);
  hipMalloc((void**) &d_b, sizeof(int) * elements);
  hipMalloc((void**) &d_prod, sizeof(int));

  *h_prod = 0;

  err = hipMemcpy(d_a, h_a, sizeof(int) * elements, hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    std::cout << "Error: " << err << std::endl;
    return;
  }

  hipMemcpy(d_b, h_b, sizeof(int) * elements, hipMemcpyHostToDevice);
  hipMemcpy(d_prod, h_prod, sizeof(int), hipMemcpyHostToDevice);

  int blocks = (elements + THREADS - 1) / THREADS;

  hipEventRecord(start);
  stride_prod<<<blocks, THREADS>>>(d_a, d_b, d_prod, elements);
  hipEventRecord(stop);
  err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cout << "Cuda last error: " << err << std::endl;
    return;
  }

  err = hipMemcpy(h_prod, d_prod, sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
    std::cout << "Error prod: " << err << std::endl;
    return;
  }

  std::cout << "Device prod: " << *h_prod << std::endl;

  err = hipEventSynchronize(stop);
  if (err != hipSuccess)
  {
    std::cout << "Cuda synchronize error: " << err << std::endl;
    return;
  }
  
  float computationTime = 0;
  hipEventElapsedTime(&computationTime, start, stop);

  std::cout << "CUDA Computation time: " << computationTime << std::endl;


  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_a);
  hipFree(d_b);
  delete(h_a);
  delete(h_b);
}
