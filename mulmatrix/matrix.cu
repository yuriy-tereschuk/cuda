#include "hip/hip_runtime.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include "matrix.h"

__global__
void IncrementRow(const int* a, int* b, int m, int k)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  b[idx] = a[idx] * 2;
}

__global__
void IncrementCol(const int* a, int* b, int m, int k)
{
  int idx = threadIdx.x + threadIdx.y;

  b[idx] = a[idx] * 2;
}

void increment_col(int* matrix_a, int* matrix_b, int n, int k)
{
  hipError_t errors;
  int *d_ma, *d_mb;

  int matrix_size = sizeof(int) * n * k;

  errors = hipMalloc(&d_ma, matrix_size);
  if (errors != hipSuccess)
  {
    std::cout << "Can't allocate memory on device for matrix A" << std::endl;
    return;
  }

  errors = hipMalloc(&d_mb, matrix_size);
  if (errors != hipSuccess)
  {
    std::cout << "Can't allocate memory on device for matrix B" << std::endl;
    return;
  }

  hipMemcpy(d_ma, matrix_a, matrix_size, hipMemcpyHostToDevice);

  IncrementCol<<<1, 16>>>(d_ma, d_mb, n, k);

  errors = hipDeviceSynchronize();
  if (errors != hipSuccess)
  {
    std::cout << "Can't start cuda computation!" << std::endl;
  }


  hipMemcpy(matrix_b, d_mb, matrix_size, hipMemcpyDeviceToHost);

  hipFree(d_ma);
  hipFree(d_mb);
}

void increment_row(int* matrix_a, int* matrix_b, int m, int k)
{
  hipError_t errors;
  int *d_ma, *d_mb;

  int matrix_size = sizeof(int) * m * k;

  errors = hipMalloc(&d_ma, matrix_size);
  if (errors != hipSuccess)
  {
    std::cout << "Can't allocate memory on device for matrix A" << std::endl;
    return;
  }

  errors = hipMalloc(&d_mb, matrix_size);
  if (errors != hipSuccess)
  {
    std::cout << "Can't allocate memory on device for matrix B" << std::endl;
    return;
  }

  hipMemcpy(d_ma, matrix_a, matrix_size, hipMemcpyHostToDevice);

  IncrementRow<<<1, 16>>>(d_ma, d_mb, m, k);

  errors = hipDeviceSynchronize();
  if (errors != hipSuccess)
  {
    std::cout << "Can't start cuda computation!" << std::endl;
  }


  hipMemcpy(matrix_b, d_mb, matrix_size, hipMemcpyDeviceToHost);

  hipFree(d_ma);
  hipFree(d_mb);
}
