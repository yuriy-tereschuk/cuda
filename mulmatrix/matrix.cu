#include "hip/hip_runtime.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include "matrix.h"

__global__
void IncrementThreads(const int* a, int* b, int m, int k)
{
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int ty = blockIdx.y * blockDim.y + threadIdx.y;

  int idx = tx + ty * m;

  b[idx] = a[idx] * threadIdx.x;
}

__global__
void IncrementBlocks(const int* a, int* b, int m, int k)
{
  int tx = blockIdx.y * blockDim.y + threadIdx.y;
  int ty = blockIdx.y * blockDim.y + threadIdx.y;

  int idx = tx + ty * m;

  b[idx] = a[idx] * threadIdx.y;
}

__global__
void MatrixTransposition(const int* a, const int* b, int* c, int m, int n, int k)
{
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int ty = blockIdx.y * blockDim.y + threadIdx.y;

  int idx = tx + ty * m;
  int idy = ty + tx * n;

  c[idy] = b[idx];
}

void matrix_transposition(int* matrix_a, int* matrix_b, int* matrix_c, int m, int n, int k)
{
  hipError_t error;
  int *d_ma, *d_mb, *d_mc;

  hipMalloc(&d_ma, sizeof(int) * k * m);
  hipMalloc(&d_mb, sizeof(int) * n * k);
  hipMalloc(&d_mc, sizeof(int) * m * n);

  error = hipMemcpy(d_ma, matrix_a, sizeof(int) * k * m, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    std::cout << "Can't init device memory for entry A." << std::endl;
    return;
  }
  
  error = hipMemcpy(d_mb, matrix_b, sizeof(int) * n * k, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    std::cout << "Can't init device memory for entry B." << std::endl;
    return;
  }

  dim3 threads(m, n);
  dim3 blocks(m/threads.x, n/threads.y);

  MatrixTransposition<<<blocks, threads>>>(d_ma, d_mb, d_mc, m, n, k);
  
  error = hipDeviceSynchronize();
  if (error != hipSuccess)
  {
    std::cout << "Can't start CUDA computations!" << std::endl;
    return;
  }

  error = hipMemcpy(matrix_c, d_mc, sizeof(int) * m * n, hipMemcpyDeviceToHost);
  if (error != hipSuccess)
  {
    std::cout << "Can't copy from device memory." << std::endl;
  }

  hipFree(d_ma);
  hipFree(d_mb);
  hipFree(d_mc);
}

void increment_threads(int* matrix_a, int* matrix_b, int n, int k)
{
  hipError_t error;
  int *d_ma, *d_mb;

  int matrix_size = sizeof(int) * n * k;

  error = hipMalloc(&d_ma, matrix_size);
  if (error != hipSuccess)
  {
    std::cout << "Can't allocate memory on device for matrix A" << std::endl;
    return;
  }

  error = hipMalloc(&d_mb, matrix_size);
  if (error != hipSuccess)
  {
    std::cout << "Can't allocate memory on device for matrix B" << std::endl;
    return;
  }

  hipMemcpy(d_ma, matrix_a, matrix_size, hipMemcpyHostToDevice);

  IncrementThreads<<<1, n*k>>>(d_ma, d_mb, n, k);

  error = hipDeviceSynchronize();
  if (error != hipSuccess)
  {
    std::cout << "Can't start cuda computation!" << std::endl;
  }

  hipMemcpy(matrix_b, d_mb, matrix_size, hipMemcpyDeviceToHost);

  hipFree(d_ma);
  hipFree(d_mb);
}

void increment_blocks(int* matrix_a, int* matrix_b, int m, int k)
{
  hipError_t error;
  int *d_ma, *d_mb;

  int matrix_size = sizeof(int) * m * k;

  error = hipMalloc(&d_ma, matrix_size);
  if (error != hipSuccess)
  {
    std::cout << "Can't allocate memory on device for matrix A" << std::endl;
    return;
  }

  error = hipMalloc(&d_mb, matrix_size);
  if (error != hipSuccess)
  {
    std::cout << "Can't allocate memory on device for matrix B" << std::endl;
    return;
  }

  hipMemcpy(d_ma, matrix_a, matrix_size, hipMemcpyHostToDevice);

  IncrementBlocks<<<m*k, 1>>>(d_ma, d_mb, m, k);

  error = hipDeviceSynchronize();
  if (error != hipSuccess)
  {
    std::cout << "Can't start cuda computation!" << std::endl;
  }


  hipMemcpy(matrix_b, d_mb, matrix_size, hipMemcpyDeviceToHost);

  hipFree(d_ma);
  hipFree(d_mb);
}
